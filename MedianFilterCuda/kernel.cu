#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "inc/helper_image.h"
#include <stdio.h>
#include <iostream>
#include <string>

#define im_size 512

__global__ void median_kernel(unsigned char *in, unsigned char *out, int filtersize)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int index = im_size*y + x;
	out[ index ] = 255;
	//printf ( "%d\n", index );
}

hipError_t median_filter_gpu ( std::string, std::string, unsigned int );

int main()
{
	/* perform median filter with GPU */
    hipError_t cudaStatus = median_filter_gpu("lena512.pgm","out.pgm", 7);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "median calculation failed!");
        return 1;
    }

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

/* wrap the kernel call here */
hipError_t median_filter_gpu(std::string inputfilename, std::string outputfilename, unsigned int size)
{
	unsigned char * host_lena = NULL;
    unsigned char * dev_input = 0;
    unsigned char * dev_output = 0;
    hipError_t cudaStatus;
	
	/* boilerplate malloc code as seen in the CUDA code */
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

	/* load up lena, allocates memory if not given */
	unsigned int width;
	unsigned int height;
	sdkLoadPGM<unsigned char> ( inputfilename.c_str(), &host_lena, &width, &height );
	
	/* create space on card for lena IN */
	cudaStatus = hipMalloc ( ( void** )&dev_input, im_size * im_size * sizeof ( unsigned char ) );
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	/* create space on card for lena OUT */
	cudaStatus = hipMalloc ( ( void** )&dev_output, im_size * im_size * sizeof ( unsigned char ) );
	if ( cudaStatus != hipSuccess )
	{
		fprintf ( stderr, "hipMalloc failed!" );
		goto Error;
	}

	/* copy host lena into card space */
	cudaStatus = hipMemcpy ( dev_input, host_lena, im_size * im_size * sizeof ( unsigned char ), hipMemcpyHostToDevice );
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	/* define kernel parameters */
	dim3 threadsPerBlock ( 16 );
	dim3 numBlocks ( im_size / threadsPerBlock.x, im_size / threadsPerBlock.y );

    /* Launch a kernel on the GPU with 32 threads for each block */
    median_kernel<<<numBlocks, threadsPerBlock>>>(dev_input, dev_output, size);

	/* check what went wrong */
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "median_kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
	/* finish up */
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching median_kernel!\n", cudaStatus);
        goto Error;
    }

	/* copy the data off */
	memset ( host_lena, 0, im_size*im_size );
	cudaStatus = hipMemcpy ( host_lena, dev_output, im_size*im_size * sizeof ( unsigned char ), hipMemcpyDeviceToHost );
	if ( cudaStatus != hipSuccess )
	{
		fprintf ( stderr, "hipMemcpy failed!" );
		goto Error;
	}

	sdkSavePGM ( outputfilename.c_str (), host_lena, width, height );

Error:
	free ( host_lena ); host_lena = NULL;
    hipFree(dev_input);
    hipFree(dev_output);
    
    return cudaStatus;
}
