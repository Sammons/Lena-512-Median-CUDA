#include "common.cuh"
#include "sobel.cuh"
#include "cpu_functions.cu"
/* prototype for call below that wraps launching the sobel filter kernel */
inline void sobel_filter ( const std::string in, const std::string out);

int main (int argc, char* argv[])
{
	if ( argc != 3 )
	{
		std::cout << "Incorrect usage, execute with parameters: <input 512x512 .pgm image path> <output path>" << std::endl;
		return 1;
	}

	std::string input_file_path = std::string ( argv[ 1 ] );
	std::string outpu_file_path = std::string ( argv[ 2 ] );
	
	/* perform sobel filter with GPU */
	sobel_filter ( input_file_path, outpu_file_path );

    return 0;
}

/* wrap the kernel call here */
/*and there is a string map that correlates an input to the correct template function to execute */
void sobel_filter ( std::string inputfilename, std::string outputfilename )
{
	unsigned char * host_lena = NULL;
	unsigned char * dev_input = 0;
	unsigned char * dev_output = 0;

	hipSetDevice ( 0 );

	/* load up lena, allocates memory if not given */
	unsigned int width;
	unsigned int height;
	sdkLoadPGM<unsigned char> ( inputfilename.c_str (), &host_lena, &width, &height );

	start ( "gpu timer" );

	/* create space on card for lena IN */
	hipMalloc ( ( void** )&dev_input, IMAGE_SIZE * IMAGE_SIZE * sizeof ( unsigned char ) );

	/* create space on card for lena OUT */
	hipMalloc ( ( void** )&dev_output, IMAGE_SIZE * IMAGE_SIZE * sizeof ( unsigned char ) );

	/* copy host lena into card space */
	hipMemcpy ( dev_input, host_lena, IMAGE_SIZE * IMAGE_SIZE * sizeof ( unsigned char ), hipMemcpyHostToDevice );

	/* define kernel parameters */
	dim3 threadsPerBlock ( 32 );
	dim3 numBlocks ( IMAGE_SIZE / threadsPerBlock.x, IMAGE_SIZE / threadsPerBlock.y );

	/* Launch a kernel on the GPU with 32 threads for each block */
	sobel_kernel <<<numBlocks, threadsPerBlock >>>( dev_input, dev_output );

	/* finish up */
	hipError_t error = hipDeviceSynchronize ();

	error = hipGetLastError ();

	/* copy the data off */
	unsigned char out[ IMAGE_SIZE*IMAGE_SIZE ] = { 0 };
	hipMemcpy ( out, dev_output, IMAGE_SIZE*IMAGE_SIZE * sizeof ( unsigned char ), hipMemcpyDeviceToHost );

	auto time = get_time ( "gpu timer" );
	std::cout << "Time: " << time << std::endl;
/*
	float accuracy = calculate_accuracy ( &out[0], host_lena, size );
	std::cout << "Accuracy: " << accuracy*100 << "% of pixels were correct" << std::endl;*/
	/* save output file */
	sdkSavePGM ( outputfilename.c_str (), out, width, height );

	/* cleanup */
	free ( host_lena ); host_lena = NULL;
	hipFree ( dev_input );
	hipFree ( dev_output );

	hipDeviceReset ();
}
