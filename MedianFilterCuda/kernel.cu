#include "common.cuh"
#include "median.cuh"
#include "median.h"

/* prototype for call below that wraps launching the median filter kernel */
hipError_t median_filter_gpu ( std::string in, std::string out, std::string size);

int main()
{

	std::string in_file = "lena.pgm", out_file = "out.pgm", size = "3";
	/* perform median filter with GPU */
    hipError_t cudaStatus = median_filter_gpu(in_file, out_file, size);
    
	/* clear the device */
	cudaStatus = hipDeviceReset();

    return 0;
}

/* wrap the kernel call here */
hipError_t median_filter_gpu(std::string inputfilename, std::string outputfilename, std::string size)
{
	unsigned char * host_lena = NULL;
    unsigned char * dev_input = 0;
    unsigned char * dev_output = 0;
    hipError_t cudaStatus;
	
    cudaStatus = hipSetDevice(0);

	/* load up lena, allocates memory if not given */
	unsigned int width;
	unsigned int height;
	sdkLoadPGM<unsigned char> ( inputfilename.c_str(), &host_lena, &width, &height );
	
	/* create space on card for lena IN */
	cudaStatus = hipMalloc ( ( void** )&dev_input, IMAGE_SIZE * IMAGE_SIZE * sizeof ( unsigned char ) );

	/* create space on card for lena OUT */
	cudaStatus = hipMalloc ( ( void** )&dev_output, IMAGE_SIZE * IMAGE_SIZE * sizeof ( unsigned char ) );

	/* copy host lena into card space */
	cudaStatus = hipMemcpy ( dev_input, host_lena, IMAGE_SIZE * IMAGE_SIZE * sizeof ( unsigned char ), hipMemcpyHostToDevice );

	/* define kernel parameters */
	dim3 threadsPerBlock ( 16 );
	dim3 numBlocks ( IMAGE_SIZE / threadsPerBlock.x, IMAGE_SIZE / threadsPerBlock.y );

    /* Launch a kernel on the GPU with 32 threads for each block */
    get_median_kernel(size) <<<numBlocks, threadsPerBlock>>>(dev_input, dev_output);

	/* check what went wrong */
    cudaStatus = hipGetLastError();
    
	/* finish up */
    cudaStatus = hipDeviceSynchronize();

	/* copy the data off */
	memset ( host_lena, 0, IMAGE_SIZE*IMAGE_SIZE );
	//cudaStatus = hipMemcpy ( host_lena, dev_output, IMAGE_SIZE*IMAGE_SIZE * sizeof ( unsigned char ), hipMemcpyDeviceToHost );

	sdkSavePGM ( outputfilename.c_str (), host_lena, width, height );

	/* cleanup */
	free ( host_lena ); host_lena = NULL;
    hipFree(dev_input);
    hipFree(dev_output);
    
    return cudaStatus;
}
