#include "hip/hip_runtime.h"
#include "common.cuh"
#include "median.cuh"


__global__ void sobel_kernel ( unsigned char *in, unsigned char *out )
{
	int x_index, y_index, pixel_index;
	set_indices ( x_index, y_index, pixel_index );

	const int x = x_index;
	const int y = y_index;

	const double kernel_x[3][3] = { { -1, 0, 1 }, { -2, 0, 2 }, { -1, 0, 1 } };
	const double kernel_y[3][3] = { { -1, -2, -1 }, { 0, 0, 0 }, { 1, 2, 1 } };

	double magnitude_x = 0.0;
	double magnitude_y = 0.0;
	for ( int i = 0; i < 3; i++ )
	{
		const int x_local = i + x;
		for ( int j = 0; j < 3; j++ )
		{
			const int y_local = j + y;
			const int index = x_local + y_local * IMAGE_SIZE;
			magnitude_x += in[ index ] * kernel_x[ i ][ j ];
			magnitude_y += in[ index ] * kernel_y[ i ][ j ];
		}
	}
	out[ x + y*IMAGE_SIZE ] = sqrt( magnitude_x*magnitude_x + magnitude_y*magnitude_y );
}


/* prototype for call below that wraps launching the median filter kernel */
hipError_t median_filter_gpu ( std::string in, std::string out, std::string size);


int main()
{

	std::string in_file = "lena.pgm", out_file = "out.pgm", size = "3";
	/* perform median filter with GPU */
    hipError_t cudaStatus = median_filter_gpu(in_file, out_file, size);
    
	/* clear the device */
	cudaStatus = hipDeviceReset();

    return 0;
}

/* wrap the kernel call here */
hipError_t median_filter_gpu(std::string inputfilename, std::string outputfilename, std::string size)
{
	unsigned char * host_lena = NULL;
    unsigned char * dev_input = 0;
    unsigned char * dev_output = 0;
    hipError_t cudaStatus;
	
    cudaStatus = hipSetDevice(0);

	/* load up lena, allocates memory if not given */
	unsigned int width;
	unsigned int height;
	sdkLoadPGM<unsigned char> ( inputfilename.c_str(), &host_lena, &width, &height );
	
	/* create space on card for lena IN */
	cudaStatus = hipMalloc ( ( void** )&dev_input, IMAGE_SIZE * IMAGE_SIZE * sizeof ( unsigned char ) );

	/* create space on card for lena OUT */
	cudaStatus = hipMalloc ( ( void** )&dev_output, IMAGE_SIZE * IMAGE_SIZE * sizeof ( unsigned char ) );

	/* copy host lena into card space */
	cudaStatus = hipMemcpy ( dev_input, host_lena, IMAGE_SIZE * IMAGE_SIZE * sizeof ( unsigned char ), hipMemcpyHostToDevice );

	/* define kernel parameters */
	dim3 threadsPerBlock ( 16 );
	dim3 numBlocks ( IMAGE_SIZE / threadsPerBlock.x, IMAGE_SIZE / threadsPerBlock.y );

    /* Launch a kernel on the GPU with 32 threads for each block */
    get_median_kernel(size) <<<numBlocks, threadsPerBlock>>>(dev_input, dev_output);

	/* check what went wrong */
    cudaStatus = hipGetLastError();
    
	/* finish up */
    cudaStatus = hipDeviceSynchronize();

	/* copy the data off */
	memset ( host_lena, 0, IMAGE_SIZE*IMAGE_SIZE );
	cudaStatus = hipMemcpy ( host_lena, dev_output, IMAGE_SIZE*IMAGE_SIZE * sizeof ( unsigned char ), hipMemcpyDeviceToHost );

	sdkSavePGM ( outputfilename.c_str (), host_lena, width, height );

	/* cleanup */
	free ( host_lena ); host_lena = NULL;
    hipFree(dev_input);
    hipFree(dev_output);
    
    return cudaStatus;
}
